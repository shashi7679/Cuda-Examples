#include "hip/hip_runtime.h"
﻿//for __syncthreads()
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include "hip/hip_runtime.h"
#include ""
#define N 512
#include <stdio.h>
#include<stdlib.h>

__global__ void check_dim() {
	printf("\n BlockIdx.x : %d   BlockIdx.y : %d   BlockIdx.z : %d",blockIdx.x,blockIdx.y,blockIdx.z);
}
__global__ void Array_Initialization(int* arr) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	arr[id] = id;
}

int main() {
	hipEvent_t start, stop;
	clock_t cpuStart, cpuEnd;
	dim3 grid(N, 1, 1);
	dim3 block(N, 1, 1);
	int* arr;
	double time_spent = 0.0;
	arr = (int*)malloc(sizeof(int) * N * N);
	cpuStart = clock();
	for (int i = 0;i < N;i++) {
		for (int j = 0;j < N;j++) {
			arr[i*N + j] = i * N + j;
		}
	}
	cpuEnd = clock();
	time_spent = (double)(cpuEnd - cpuStart) / CLOCKS_PER_SEC;
	printf("Time taken in CPU initalization : %lf \n",time_spent);
	int* d;
	hipMalloc(&d, sizeof(int) * N * N);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);
	Array_Initialization << <grid, block >> > (d);
	hipMemcpy(arr, d, sizeof(int) * N * N, hipMemcpyDeviceToHost);
	hipEventRecord(stop);
	
	hipEventSynchronize(stop);
	float milli = 0.0;
	hipEventElapsedTime(&milli, start, stop);
	printf("Time taken in GPU initalization : %f \n", milli);
	/*for (int i = 0;i < N;i++) {
		for (int j = 0;j < N;j++) {
			printf(" %d ", arr[j + i * N]);
		}
		printf("\n");
	}*/
	//check_dim << <block,1 >> > ();
	return 0;
}