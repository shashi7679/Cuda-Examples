#include "hip/hip_runtime.h"
﻿/*You are given a 1024x1024 matrix A where each thread is handling A[i][j]. 
Each row of the matrix is assigned to one block of the GPU. 
Use shared memory to update A[i][j] to A[i][j+1].
*/
//for __syncthreads()
#ifndef __HIPCC__ 
#define __HIPCC__
#endif
#include "hip/hip_runtime.h"
#include ""
#define N 32
#include <stdio.h>
#include<stdlib.h>

/*__global__ void check_dim() {
	printf("\n BlockIdx.x : %d   BlockIdx.y : %d   BlockIdx.z : %d",blockIdx.x,blockIdx.y,blockIdx.z);
}*/

__global__ void Array_Initialization(int* arr) {
	int id = N*threadIdx.y + threadIdx.x + N*N*blockIdx.x + N*N*N*blockIdx.y;
	arr[id] = id;
	//printf("\n id : %d    arr[id] = %d ", id, arr[id]);
}
__global__ void Matx_Manipulate(int* arr) {
	__shared__ int s[N*N];
	int id = N * threadIdx.y + threadIdx.x + N * N * blockIdx.x + N * N * N * blockIdx.y;
	int id_perblock = threadIdx.x + N * threadIdx.y;
	if (id_perblock == 1023) s[id_perblock] = 1024 * (blockIdx.x + blockIdx.y * N + 1);
	else s[id_perblock] = arr[id + 1];
	__syncthreads();
	arr[id] = s[id_perblock];
	//printf("\n id : %d    arr[id] = %d ", id, arr[id]);
}
int main() {
	hipEvent_t start, stop;
	//clock_t cpuStart, cpuEnd;
	dim3 grid(N, N, 1);
	dim3 block(N, N, 1);
	int* arr;
	int* arr_mod;
	double time_spent = 0.0;
	arr = (int*)malloc(sizeof(int) * N * N* N * N);
	arr_mod = (int*)malloc(sizeof(int) * N * N * N * N);
	//cpuStart = clock();
	/*for (int i = 0;i < N;i++) {
		for (int j = 0;j < N;j++) {
			arr[i*N + j] = i * N + j;
		}
	}
	cpuEnd = clock();
	time_spent = (double)(cpuEnd - cpuStart) / CLOCKS_PER_SEC;
	printf("Time taken in CPU initalization : %lf \n",time_spent);*/
	int* d;
	hipMalloc(&d, sizeof(int) * N * N* N * N);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);
	Array_Initialization << <grid, block >> > (d);
	hipMemcpy(arr, d, sizeof(int) * N * N * N * N, hipMemcpyDeviceToHost);
	hipEventRecord(stop);
	
	hipEventSynchronize(stop);
	float milli = 0.0;
	hipEventElapsedTime(&milli, start, stop);
	printf("Time taken in GPU initalization : %f \n", milli);
	printf("\n Before Modification :- ");
	for (int i = 0;i < N*N;i++) {
		for (int j = 0;j < N * N;j++) {
			printf(" %d ", arr[i*N*N + j]);
		}
		printf("\n");
	}
	Matx_Manipulate << <grid, block >> > (d);
	hipMemcpy(arr_mod, d, sizeof(int) * N * N * N * N, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	printf("\n After Modification :- ");
	for (int i = 0;i < N * N;i++) {
		for (int j = 0;j < N * N;j++) {
			printf(" %d ", arr_mod[i * N * N + j]);
		}
		printf("\n");
	}
	return 0;
}